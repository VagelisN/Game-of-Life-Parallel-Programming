#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void kernel_func(int *ccurrent, int *cnext,int arraysize)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;// get the row number
	int col = blockIdx.x * blockDim.x + threadIdx.x;// get the col number
	if(row >= arraysize || col >= arraysize)return;// check if the dimensions are over the dims of the array
	/*Because the ccurent array is a flattened 2d array we access each element ccurent[(row * arraysize) + column]*/
	int neighbours = 0;//start checking the neighbours
	if((col-1) < 0){if(ccurrent[row * arraysize + (arraysize -1)] == 1 )neighbours++;}//West neighbour
    else if(ccurrent[row * arraysize +(col-1)] == 1 )neighbours++;//west

    if((row+1) > (arraysize-1))//South-West neighbour
    {
     	if((col-1) < 0){if(ccurrent[(arraysize-1)] == 1)neighbours++;}
      	else if(ccurrent[(col-1)] == 1)neighbours++;
    }
    else if((col-1) < 0){if(ccurrent[(row+1) * arraysize + (arraysize-1)] == 1)neighbours++;}//sw
    else if(ccurrent[(row+1) * arraysize +(col-1)] == 1 )neighbours++;//sw
                
    if((row+1) > arraysize-1){if(ccurrent[col] == 1)neighbours++;}//South neighbour
    else if(ccurrent[(row+1) * arraysize + col] == 1 )neighbours++;//s
                
    if((row+1) > (arraysize-1))//South- East neighbour
    {
        if((col+1) > (arraysize-1)){if(ccurrent[0] == 1)neighbours++;}
    	else if(ccurrent[col+1] == 1)neighbours++;
    }
    else if((col+1) > (arraysize-1)){if(ccurrent[(row+1) * arraysize] == 1)neighbours++;}//se
    else if(ccurrent[(row+1) * arraysize + col+1] == 1 )neighbours++;//se
                
    if((col+1) > (arraysize-1)){if(ccurrent[row * arraysize] == 1)neighbours++;}//East neighbour
    else if(ccurrent[row * arraysize + col+1] == 1 )neighbours++;//e

    if((row-1) < 0)//North- East neighbour
    {
    	if((col+1) > (arraysize-1) ){if(ccurrent[(arraysize-1) * arraysize] == 1)neighbours++;}
        else if(ccurrent[(arraysize-1)* arraysize + col+1] == 1)neighbours++;
    }
    else if((col+1) > arraysize-1){if(ccurrent[(row-1) * arraysize] == 1)neighbours++;}//ne
    else if(ccurrent[(row-1) * arraysize + col+1] == 1 )neighbours++;//ne
                
    if((row-1) < 0){if(ccurrent[(arraysize-1)* arraysize + col])neighbours++;}//North neighbour
    else if(ccurrent[(row-1) * arraysize + col] == 1) neighbours++;//n
                
    if((row-1) < 0)//North- West neighbour
    {
    	if((col-1) < 0 ){if(ccurrent[(arraysize-1) * arraysize + arraysize-1] == 1)neighbours++;}
        else if(ccurrent[(arraysize-1) * arraysize + col-1] == 1)neighbours++;
    }
    else if((col-1) < 0){if(ccurrent[(row-1) * arraysize + arraysize-1] == 1)neighbours++;}//nw
    else if(ccurrent[(row-1) * arraysize + col-1] == 1 )neighbours++;//nw
                   
    if(ccurrent[row * arraysize +col] == 0)
    {
    	if(neighbours == 3)cnext[row * arraysize + col]=1;//If the array[row][col] = 1 check the number of neighbours and set the next value 
        else cnext[row * arraysize + col] = 0;
    }
    if(ccurrent[row * arraysize + col] == 1)
    {
    	if(neighbours>1 && neighbours<4)cnext[row * arraysize + col] = 1;
    	else cnext[row * arraysize + col] = 0;
    }
}


__global__ void kernel_checksimilarity(int *ccurrent, int *cnext, int arraysize, int *dsame, int *ddead)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;// get the row number
	int col = blockIdx.x * blockDim.x + threadIdx.x;// get the col number
	if(row >= arraysize || col >= arraysize)return;// check if the dimensions are over the dims of the array
	if(ccurrent[row * arraysize + col] != cnext[row * arraysize + col]) *dsame = 0;
	if(cnext[row * arraysize + col] == 1)*ddead = 0;
}



int main(int argc, char *argv[])
{
	srand(time(NULL));
	unsigned int i, j, arraysize = 50,check = 31;
	int *same , *dead;
	int *dsame = NULL, *ddead = NULL;
	same = (int*)malloc(sizeof(int));
	dead = (int*)malloc(sizeof(int));
	hipMalloc(&dsame,sizeof(int));
	hipMalloc(&ddead,sizeof(int));
	unsigned long generations = 50;
	char c;
	while((c = getopt (argc, argv, "g:s:")) != -1)
	{
		switch(c)
		{
			case 'g':
				generations = atoi(optarg);
				break;
			case 's':
				arraysize = atoi(optarg);
				break;
			default:
				break;
		}		
	}
	//Host mallocs and initialization of the arrays
	int *currentdata = (int*)malloc(arraysize * arraysize * sizeof(int));
	int **current = (int**)malloc(arraysize * sizeof(int*));
	int *nextdata = (int*)malloc(arraysize *arraysize * sizeof(int));
	int **next = (int**)malloc(arraysize * sizeof(int*));
	for (i = 0; i < arraysize; ++i)
	{
		current[i] = &(currentdata[i*arraysize]);
		next[i] = &(nextdata[i*arraysize]);
	}
	for (i = 0; i < arraysize; ++i)
	{
		for (j= 0; j < arraysize; ++j)
		{
			current[i][j] = rand()%2;
			next[i][j] = 0;
			//printf("%d ", current[i][j]);
		}
		//printf("\n");
	}
	//Device mallocs and transfer HostToDevice
	int *ccurrent = NULL;
	hipMalloc(&ccurrent,(arraysize * arraysize * sizeof(int)));
	int *cnext = NULL;
	hipMalloc(&cnext,(arraysize * arraysize * sizeof(int)));
	hipMemcpy(ccurrent, currentdata, (arraysize * arraysize * sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(cnext, nextdata, (arraysize * arraysize * sizeof(int)), hipMemcpyHostToDevice);
	int *ctemp;
	//dim3 dimBlock(block_size, block_size);
	//dim3 dimGrid((arraysize + dimBlock.x - 1) / dimBlock.x,(arraysize + dimBlock.y - 1) / dimBlock.y);
	
	hipDeviceProp_t myCUDA;
	if (hipGetDeviceProperties(&myCUDA, 0) == hipSuccess)
	{
	    printf("Using device %d: ", 0);
	    printf("%s; global mem: %dΒ; compute v%d.%d; clock: %d kHz\n",myCUDA.name, (int)myCUDA.totalGlobalMem, (int)myCUDA.major,(int)myCUDA.minor, (int)myCUDA.clockRate);
	}
	int threadsPerBlock = myCUDA.maxThreadsPerBlock;
	int temp = (int)sqrt(threadsPerBlock);
	printf("Maximum threads per block dimension = %d\n",temp);
	dim3 dimBlock(temp, temp);
	dim3 dimGrid((arraysize + dimBlock.x - 1) / dimBlock.x,(arraysize + dimBlock.y - 1) / dimBlock.y);
	float ttime;
	hipEvent_t start, stop;//Start the clock

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (i = 0; i < generations; ++i)
	{
		kernel_func<<<dimGrid, dimBlock>>>(ccurrent,cnext,arraysize);//call the Device function
		hipDeviceSynchronize();
		if((i%check) == 0)
		{
			*same = 1;
			*dead = 1;
			hipMemcpy(dsame, same, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(ddead, dead, sizeof(int), hipMemcpyHostToDevice);
			kernel_checksimilarity<<<dimGrid,dimBlock>>>(ccurrent,cnext,arraysize,dsame,ddead);
			hipDeviceSynchronize();
			hipMemcpy(same, dsame, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(dead, ddead, sizeof(int), hipMemcpyDeviceToHost);
			if((*same == 1) || (*dead == 1))break;
		}
		ctemp = ccurrent;//Change the arrays for the next generation
		ccurrent = cnext;
		cnext = ctemp;
		/*hipMemcpy(currentdata, ccurrent, (arraysize * arraysize * sizeof(int)), hipMemcpyDeviceToHost);//DevicetoHost transfer
		printf("In generation %d\n",i);
		for ( k = 0; k < arraysize; ++k)//print the array
		{
			for (j = 0; j < arraysize; ++j)
			{
				printf("%d ",current[k][j]);
			}
			printf("\n");
		}*/
	}
	hipEventRecord(stop, 0);//stop the clock
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ttime, start, stop);
	printf("\nProcess exited at generation: %d with arraysize: %d x %d, after:  %3.1f ms \n", i, arraysize, arraysize,ttime);
	if(*dead == 1)printf("Exited earlier because all cells are dead.\n");
	if(*same == 1)printf("Exited earlier because current and next generations are the same.\n");
	/*printf("Last version of the array\n");
	hipMemcpy(currentdata, ccurrent, (arraysize * arraysize * sizeof(int)), hipMemcpyDeviceToHost);
	for ( i = 0; i < arraysize; ++i)
	{
		for (j = 0; j < arraysize; ++j)
		{
			printf("%d ",current[i][j]);
		}
		printf("\n");
	}*/
	//Free the allocated memory

	hipFree(ccurrent);
	hipFree(cnext);
	hipFree(dsame);
	hipFree(ddead);
	free(same);
	free(dead);
	free(nextdata);
	free(next);
	free(currentdata);
	free(current);
	return 0;
}